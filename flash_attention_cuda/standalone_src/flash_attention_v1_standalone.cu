#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(condition)                                                  \
  do {                                                                         \
    hipError_t error = condition;                                             \
    if (error != hipSuccess) {                                                \
      printf("CUDA_CHECK error in line %d of file %s \
              : %s \n",                                                        \
             __LINE__, __FILE__, hipGetErrorString(hipGetLastError()));      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define DEBUG

#ifdef DEBUG
#define DEBUG_BLOCK(expr)                                                      \
  do {                                                                         \
    expr                                                                       \
  } while (0)
#else
#define DEBUG_BLOCK(...)                                                       \
  do {                                                                         \
  } while (0)
#endif


// data type to test
using FP = float;
// BLOCK_M(Br, Brow), BLOCK_N(Bc, Bcol) can be determined at compile time
// just like offical implementation which use a template kernel to do that
// Block row size
const int Br = 2;
// Block column size
const int Bc = 2;
// seqlen
const int input_seq = 4;
// dim
const int dim = 4;


__global__ void naive_nrow_gemm(float *A, float *B, float *C, float a, float b,
                                int M, int N, int K, int mBlock);
__global__ void row_softmax(float *input, float *output, int n);
__global__ void naive_pv(float *P, float *V, float *O, int M, int N,
                         int mBlock);

__global__ void flash_attention_v1_kernel(FP *Q, FP* K, FP* V, FP* O, FP* gMAX, FP* gDenom, int seqlen, FP smScale);
void print_host_matrix(float *matrix, int m, int n);
void print_device_matrix(float *matrix, int m, int n);

void flash_attention_v1_cuda(FP *Q, FP *K, FP *V, FP *O, int m, int n) {
  FP *dev_max, *dev_denom, *host_max, *host_denom;
  // qk buffer
  FP *QK;

  FP sm_scale = 1.f / sqrtf(static_cast<FP>(n));
  int BS = 1;
  int HEAD = 1;
  int SEQLEN = m;
  int DIM = n;

  host_max = new FP[SEQLEN];
  host_denom = new FP[SEQLEN];
  for (int i = 0; i < SEQLEN; i++) {
    host_max[i] = -INFINITY;
    host_denom[i] = 0;
  }

  CUDA_CHECK(hipMalloc((void **)&dev_max, sizeof(FP) * SEQLEN * DIM));
  CUDA_CHECK(hipMalloc((void **)&dev_denom, sizeof(FP) * SEQLEN * DIM));
  CUDA_CHECK(hipMalloc((void **)&QK, sizeof(FP) * SEQLEN * SEQLEN));
  CUDA_CHECK(hipMemcpy(dev_max, host_max, sizeof(FP) * SEQLEN * DIM, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_denom, host_denom, sizeof(FP) * SEQLEN * DIM, hipMemcpyHostToDevice));


  int Gc = 1;
  int Gr = (SEQLEN + Br - 1) / Br;

  // NOTE: each block process a range row of Q
  dim3 grid = dim3(Gc, Gr);
  // NOTE: each thread process a tile of Q
  dim3 block = dim3(Bc, Br);
  flash_attention_v1_kernel<<<grid, block>>>(Q, K, V, O, dev_max, dev_denom, SEQLEN, sm_scale);

  printf("== V1: O ==\n");
  print_device_matrix(O, SEQLEN, DIM);

  hipFree(QK);
  hipFree(dev_max);
  hipFree(dev_denom);
}

__global__ void flash_attention_v1_kernel(FP *Q, FP* K, FP* V, FP* O, FP* gMAX, FP* gDenom, int seqlen, FP smScale) {
  // block size for K, V
  // group of row(seqlen)
  int groupSeq = (seqlen + Bc - 1) / Bc;
  // parallel process for V[Br, d]
  // group of column
  int groupTx = (dim + Bc - 1) / Bc;
  int groupTy = (dim + Br - 1) / Br;

  // load slice from global memory(HBM)
  __shared__ FP sQ[Br][dim];
  __shared__ FP sK[Bc][dim];
  __shared__ FP sV[Bc][dim];
  __shared__ FP sO[Br][dim];
  __shared__ FP sQK[Br][Bc];

  __shared__ FP sNewO[Br][dim];
  // e^{x - max}
  __shared__ FP sSafeE[Br][Bc];
  // s stand for shared and local
  __shared__ FP sDenom[Br];
  __shared__ FP sMax[Br];

  // TODO: multihead

  // [0, Bc]
  int tx = threadIdx.x;
  // [0, Br]
  int ty = threadIdx.y;

  int row = ty + blockIdx.y * blockDim.y;
  for (int j = 0; j < groupSeq; j++) {
    if ((j * Bc + tx) < seqlen) {
      // load k, v from global memory to shared memory
      // K[seqlen, dim], V[seqlen, dim]
      for (int i = 0; i < groupTy; i++) {
        // each thread.x copy a row of K to K.T
        // row0, t0:
        // row1, t1:
        // row2, t0:
        // row3, t2:
        sK[tx][i * Br + ty] = K[j * Bc * dim + tx * dim + i * Br + ty];
        sV[tx][i * Br + ty] = V[j * Bc * dim + tx * dim + i * Br + ty];
      }
    }

    if (row < seqlen) {
      // load q, o, max, denom from global memory to shared memory
      // Q[seqlen, dim]
      for (int i = 0; i < groupTx; i++) {
        sQ[ty][i * Bc + tx] = Q[row * dim + i * Bc + tx];
        sO[ty][i * Bc + tx] = O[row * dim + i * Bc + tx];
      }

      // NOTE: the drawback of flash attention 1 is here that it will load O, max, denom from global memory to shared memory many time
      sMax[ty] = gMAX[row];
      sDenom[ty] = gDenom[row];
    }

    // wait until g2s done
    __syncthreads();

    // compute qk
    FP sum = 0.f;
    // result oriented: qk[y][x] from q[y] @ k[x]
    for (int i = 0; i < dim; i++) {
      sum += sQ[ty][i] * sK[tx][i];
    }
    // sQK[Br, Bc]
    sQK[ty][tx] = sum * smScale;

    // wait until qk done
    __syncthreads();

    // compute local max of each row of qk
    FP localMax = -INFINITY;
    for (int i = 0; i < Bc; i++) {
      localMax = max(localMax, sQK[ty][i]);
    }
    __syncthreads();

    // compute safe e(e^{x - max}) of each qk element
    sSafeE[ty][tx] = exp(sQK[ty][tx] - localMax);
    __syncthreads();

    // accumulate local denom of each row of qk with local max
    FP localDenom = 0.f;
    for (int i = 0; i < Bc; i++) {
      localDenom += sSafeE[ty][i];
    }
    __syncthreads();

    // NOTE: this is a pure flash attention 1 implementation with many redundant mul
    // update global max of each row
    FP newMax = max(sMax[ty], localMax);
    // rescale history result
    FP rescaleOld = exp(sMax[ty] - newMax);
    // rescale result just computed above: sSafeE, localDenom
    FP rescaleCur = exp(localMax - newMax);
    FP newDenom = sDenom[ty] * rescaleOld + localDenom * rescaleCur;

    // clean each row of of sNewO
    for (int i = 0; i < groupTx; i++) {
      sNewO[ty][i * Bc + tx] = 0;
    }

    // NOTE: 
    // QK[Br, Bc] @ V[Bc, d] = O[Br, d]
    // tx in [0, Bc], ty in [0, Br]
    // slice-Bc and each O[ty, group.x] as accumulator
    for (int k = 0; k < Bc; k++) {
      for (int i = 0; i < groupTx; i++) {
        // rescale numerator
        sNewO[ty][i * Bc + tx] += sSafeE[ty][k] * rescaleCur * sV[k][i * Bc + tx];
      }
    }

    // NOTE: rescale output
    // old_nume = old_o * old_denom
    // new_o = (old_nume + new_nume) / new_denom
    for (int i = 0; i < groupTx; i++) {
      sNewO[ty][i * Bc + tx] = (/* new_nume */ sNewO[ty][i * Bc + tx] + /* old_o */sO[ty][i * Bc + tx] * rescaleOld * /* old_denom */ sDenom[ty]) / newDenom;
    }

    __syncthreads();

    // update global o
    if (row < seqlen) {
      for (int i = 0; i < groupTx; i++) {
        // copy sO[row, dim] to gO[row, dim]
        O[row * dim + i * Bc + tx] = sNewO[ty][i * Bc + tx];
      }
    }

    // update global max and denom
    gMAX[row] = newMax;
    gDenom[row] = newDenom;
    __syncthreads();
  }
}

void self_attention_cuda(float *Q, float *K, float *V, float *O, int m, int n) {
  int mBlock = 2;
  assert(m % mBlock == 0 && "mBlock should align");

  float sm_scale = 1.f / sqrtf(static_cast<float>(n));
  float *sm_o;
  hipMalloc((void **)&sm_o, sizeof(float) * m * m);

  dim3 qk_block(m / mBlock, 1, 1);
  naive_nrow_gemm<<<1, qk_block>>>(Q, K, sm_o, sm_scale, 0, m, m, n, mBlock);
  hipDeviceSynchronize();
  DEBUG_BLOCK(
    CUDA_CHECK(hipGetLastError());
    printf("== naive QK ==\n");
    print_device_matrix(sm_o, m, m);
  );

  // QK[M, M]
  dim3 sm_block(m, 1, 1);
  row_softmax<<<1, sm_block>>>(sm_o, sm_o, m);
  hipDeviceSynchronize();
  DEBUG_BLOCK(
    CUDA_CHECK(hipGetLastError());
    printf("== naive softmax(QK) ==\n");
    print_device_matrix(sm_o, m, m);
  );

  // QK[M, M] @ V[M, N]
  dim3 qkv_block(m / mBlock, 1, 1);
  naive_pv<<<1, qkv_block>>>(sm_o, V, O, m, n, mBlock);
  hipDeviceSynchronize();
  DEBUG_BLOCK(
    CUDA_CHECK(hipGetLastError());
    printf("== naive softmax(QK)V ==\n");
    print_device_matrix(O, m, n);
  );

  hipFree(sm_o);
}

// naive gemm implement with slice-k
// perform C = aA@B + bC
// A[M, K] x B[K, N] = C[M, N]
// each thread process mblock rows of A
__global__ void naive_nrow_gemm(float *A, float *B, float *C, float a, float b,
                                int M, int N, int K, int mBlock) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // each thread process a range of rows
  idx *= mBlock;

  // A[mBlock, K] x B[N, K].T = C[mBlock, N]
  for (int i = idx; i < idx + mBlock; i++) {
    for (int j = 0; j < N; j++) {
      float sum = 0.f;
      for (int k = 0; k < K; k++) {
        sum += A[i * K + k] * B[j * K + k];
      }
      // C[M, N]
      // C = aA@B + bC
      C[i * N + j] = a * sum + b * C[i * N + j];
    }
  }
}

// perform QK[M, M] @ V[M, N]
__global__ void naive_pv(float *P, float *V, float *O, int M, int N,
                         int mBlock) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // each thread process a range of rows
  idx *= mBlock;

  int K = M;
  // P[mBlock, M] x V[M, N] = O[mBlock, N]
  for (int i = idx; i < idx + mBlock; i++) {
    for (int j = 0; j < N; j++) {
      float sum = 0.f;
      for (int k = 0; k < K; k++) {
        sum += P[i * K + k] * V[k * N + j];
      }
      // C[M, N]
      O[i * N + j] = sum;
    }
  }
}

// each thread process one row of softmax
__global__ void row_softmax(float *input, float *output, int n) {
  // assume id will not exceed row number of input
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  float max = -INFINITY;
  float sum = 0.f;

  // Find max
  for (int i = 0; i < n; i++) {
    if (input[idx * n + i] > max) {
      max = input[idx * n + i];
    }
  }

  // Compute numerator and denominator
  for (int i = 0; i < n; i++) {
    output[idx * n + i] = exp(input[idx * n + i] - max);
    sum += output[idx * n + i];
  }

  // Compute softmax
  for (int i = 0; i < n; i++) {
    output[idx * n + i] /= sum;
  }
}

// print matrix
void print_host_matrix(float *matrix, int m, int n) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      printf("%f, ", matrix[i * n + j]);
    }
    printf("\n");
  }
}

void print_device_matrix(float *dev_ptr, int m, int n) {
  float *host_ptr = new float[m * n];
  hipMemcpy(host_ptr, dev_ptr, sizeof(float) * m * n, hipMemcpyDeviceToHost);

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      printf("%f, ", host_ptr[i * n + j]);
    }
    printf("\n");
  }
  free(host_ptr);
}

void test_attention() {
  // seqlen
  int m = input_seq;
  // dim
  int n = dim;

  // Host pointer
  float *h_K = new float[m * n];
  float *h_Q = new float[m * n];
  float *h_V = new float[m * n];
  float *h_O = new float[m * n];

  // 初始化 K, Q, V
  for (int i = 0; i < m * n; ++i) {
    // h_K[i] = static_cast<float>(rand()) / RAND_MAX;
    // h_Q[i] = static_cast<float>(rand()) / RAND_MAX;
    // h_V[i] = static_cast<float>(rand()) / RAND_MAX;
    h_K[i] = static_cast<float>(i);
    h_Q[i] = static_cast<float>(i);
    h_V[i] = static_cast<float>(i);
  }

  printf("== K ==\n");
  print_host_matrix(h_K, m, n);

  float *d_K, *d_Q, *d_V, *d_O;
  // Malloc device memory
  hipMalloc((void **)&d_K, sizeof(float) * m * n);
  hipMalloc((void **)&d_Q, sizeof(float) * m * n);
  hipMalloc((void **)&d_V, sizeof(float) * m * n);
  hipMalloc((void **)&d_O, sizeof(float) * m * n);

  // Copy data from host to device
  hipMemcpy(d_K, h_K, sizeof(float) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(d_Q, h_Q, sizeof(float) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(d_V, h_V, sizeof(float) * m * n, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Run test
  for (int i = 0; i < 1; i++) {
    // Launch kernel
    self_attention_cuda(d_Q, d_K, d_V, d_O, m, n);

    CUDA_CHECK(hipGetLastError());
  }

  // test flash attention 1
  hipMemset(d_O, 0, sizeof(float) * m * n);
  for (int i = 0; i < 1; i++) {
    flash_attention_v1_cuda(d_Q, d_K, d_V, d_O, m, n);
    CUDA_CHECK(hipGetLastError());
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for kernel execution: %.3f ms \n", milliseconds / 100);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Result back to host
  hipMemcpy(h_O, d_O, sizeof(float) * m * n, hipMemcpyDeviceToHost);

  hipFree(d_K);
  hipFree(d_Q);
  hipFree(d_V);
  hipFree(d_O);
  free(h_Q);
  free(h_K);
  free(h_V);
  free(h_O);
}

int main() {
  test_attention();

  return 0;
}

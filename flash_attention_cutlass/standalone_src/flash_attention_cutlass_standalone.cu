#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>

#include "flash.h"
#include "kernel_traits.h"
#include "utils.h"

#define DEBUG

// data type to test
using FP = float;
using FPC = cute::half_t;
// Out type
#ifdef DEBUG
// 精度测试
using FPC_O = float;
#else
using FPC_O = cute::half_t;
#endif
// using FPC = double;
// BLOCK_M(Br, Brow), BLOCK_N(Bc, Bcol) can be determined at compile time
// just like offical implementation which use a template kernel to do that
// Block row size
// TODO: 测试这里多种shape
const int Bm = 64;
// Block column size
const int Bn = 64;
// TODO: 测试更大规模, 或者warps=2

// TODO: causal模式下, warp!=1情况有bug
// 使用kNThreads
const int Warps = 4;
const bool IS_CAUSAL = false;

const int BS = 2;
const int HEAD = 16;
const int SEQLEN = 128 * 3;
const int DIM = 64;
// const float softmax_scale = 1.f / sqrtf(static_cast<float>(SEQLEN));
const float softmax_scale = 1.f;

// debug only
int TX = 3;
int TY = 0;

// TODO: test trait
using Test_Traits = Flash_fwd_kernel_traits<DIM, Bm, Bn, Warps, FPC>;


// Shared Storage with Aligned addresses.
template <class ElementType, class SmemLayoutQ, class SmemLayoutK, class SmemLayoutV>
struct SharedStorage {
  // TODO: Aligned的话smem的计算是否有问题
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutQ>> smem_q;
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutK>> smem_k;
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayoutV>> smem_v;
};


#define CUDA_CHECK(condition)                                                  \
  do {                                                                         \
    hipError_t error = condition;                                             \
    if (error != hipSuccess) {                                                \
      printf("CUDA_CHECK error in line %d of file %s \
              : %s \n",                                                        \
             __LINE__, __FILE__, hipGetErrorString(error));      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// #define DEBUG

#ifdef DEBUG
#define DEBUG_BLOCK(expr)                                                      \
  do {                                                                         \
    expr                                                                       \
  } while (0)
#else
#define DEBUG_BLOCK(...)                                                       \
  do {                                                                         \
  } while (0)
#endif

// TODO: 待功能差不多后再使用torch
void set_params_fprop(Flash_fwd_params &params,
                      // sizes
                      const size_t bs, const size_t head, const size_t seqlen,
                      const size_t dim,

                      const size_t bs_stride, const size_t head_stride,
                      const size_t seqlen_stride, const size_t dim_stride,

                      // // device pointers
                      // const torch::Tensor q,
                      // const torch::Tensor k,
                      // const torch::Tensor v,
                      // torch::Tensor out,

                      void *q, void *k, void *v, void *out,

                      float softmax_scale) {

  memset(&params, 0, sizeof(params));

  params.bs = bs;
  params.head = head;
  params.seqlen = seqlen;
  params.dim = dim;

  params.bs_stride = bs_stride;
  params.head_stride = head_stride;
  params.seqlen_stride = seqlen_stride;
  params.dim_stride = dim_stride;

  params.softmax_scale = softmax_scale;

  // TODO: get ptr
  params.q_ptr = q;
  params.k_ptr = k;
  params.v_ptr = v;
  params.out_ptr = out;
}

__global__ void naive_nrow_gemm(FP *A, FP *B, FP *C, FP a, FP b,
                                int M, int N, int K, int mBlock);
__global__ void causal_mask_qk(FP *qk, int m);
__global__ void row_softmax(FP *input, FP *output, int n);
__global__ void naive_pv(FP *P, FP *V, FP *O, int M, int N,
                         int mBlock);

template<typename T, typename U>
bool all_close(T *A, U *B, int total_size);

namespace flash {

using namespace cute;

template <int kBlockM, int kBlockN, int kNWarps,typename Engine, typename Layout>
inline __device__ void mask_within_nblock(Tensor<Engine, Layout> &tensor, const int m_block, const int nbi) {
    // tensor has shape (nrow=(2, MMA_M), ncol=(2, MMA_N))
    static_assert(Layout::rank == 2, "Only support 2D Tensor");

    // NOTE:
    // 确定一个MMA内的index也是一个难点
    // (nrow=(2, MMA_M), ncol=(2, MMA_N))形如:
    //    T1.V0 T1.V1
    //    T1.V0 T1.V1
    // 根据mma_tile的示意图来确定col和row值

    // NOTE:
    // 计算thread的处理范围, mask掉超出范围的部分
    //
    // NOTE:
    // % 32表示32做组, 因为SM80_16x8x16_F32F16F16F32_TN _1_2_1中最大线程数id是32
    // (lane_id % 4) * 2表示在哪个"颜色"的col(thread)中, *2是为了靠右(即处理的哪个value2)
    // 因此col_idx_offset表示当前thread所处理的单个Atom中4列的哪列

    // lane_id表示一个MMA tile中的"线程组"
    const int lane_id = threadIdx.x % 32;
    const int col_idx_offset = kBlockN * nbi + (lane_id % 4) * 2;

    const int nrow_group = threadIdx.x / 32;
    const int row_idx_offset = kBlockM * m_block + lane_id / 4 + nrow_group * 16 /* 2*8 */;
    // (2, nrow), 2*8 for each
    const int group_stride = kNWarps * 16;

    #pragma unroll
    for (int nj = 0; nj < size<1, 1>(tensor); ++nj) {
        // SM80_16x8x16_F32F16F16F32_TN中的一组中, 一行4个线程处理8个value
        const int col_idx_base = col_idx_offset + nj * 8;
        #pragma unroll
        for (int j = 0; j < size<1, 0>(tensor); ++j) {
            // j用于计算value 1和value 2对应col
            // col_idx最终表示当前thread所处理的value的列号
            const int col_idx = col_idx_base + j;

            // mask掉scores中(QK后的结果)超出范围的部分
            // 列号和行号对比

            // Without the "make_coord" we get wrong results
            // for nrow(2, MMA_M)
            #pragma unroll
            for (int mi = 0; mi < size<0, 0>(tensor); ++mi) {

              #pragma unroll
              for (int mj = 0; mj < size<0, 1>(tensor); ++mj) {
                const int row_idx = row_idx_offset + mi * 8 + mj * group_stride;
                if (col_idx > row_idx) {
                  tensor(make_coord(mi, mj), make_coord(j, nj)) = -INFINITY;
                }
              }

            }

        }
    }
}


// NOTE: A矩阵已经在寄存器中的gemm封装
template<typename Tensor0, typename Tensor1, typename Tensor2, typename Tensor3,
         typename TiledMma, typename TiledCopy, typename ThrCopy>
inline __device__ void gemm_A_in_regs(Tensor0 &acc, Tensor1 &tCrA, Tensor2 &tCrB, Tensor3 const& tCsB,
                                      TiledMma tiled_mma, TiledCopy smem_tiled_copy_B,
                                      ThrCopy smem_thr_copy_B) {
    // NOTE: 符合M N K描述: A[M, K] @ B[N, K] = C[M, N]
    CUTE_STATIC_ASSERT_V(size<1>(tCrA) == size<1>(acc));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<1>(tCrB) == size<2>(acc));                     // MMA_N
    CUTE_STATIC_ASSERT_V(size<2>(tCrA) == size<2>(tCrB));                     // MMA_K
    // NOTE: retile 成拷贝需要的大小
    Tensor tCrB_copy_view = smem_thr_copy_B.retile_D(tCrB);
    CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));            // N
    cute::copy(smem_tiled_copy_B, tCsB(_, _, _0{}), tCrB_copy_view(_, _, _0{}));
    #pragma unroll
    for (int i = 0; i < size<2>(tCrA); ++i) {
        if (i < size<2>(tCrA) - 1) {
            cute::copy(smem_tiled_copy_B, tCsB(_, _, i + 1), tCrB_copy_view(_, _, i + 1));
        }
        cute::gemm(tiled_mma, tCrA(_, _, i), tCrB(_, _, i), acc);
    }
}

template<typename Tensor0, typename Tensor1,
         typename Tensor2, typename Tensor3, typename Tensor4,
         typename TiledMma, typename TiledCopyA, typename TiledCopyB,
         typename ThrCopyA, typename ThrCopyB>
inline __device__ void gemm_smem(Tensor0 &acc, Tensor1 &tCrA, Tensor2 &tCrB, Tensor3 const& tCsA,
                            Tensor4 const& tCsB, TiledMma tiled_mma,
                            TiledCopyA smem_tiled_copy_A, TiledCopyB smem_tiled_copy_B,
                            ThrCopyA smem_thr_copy_A, ThrCopyB smem_thr_copy_B) {
    CUTE_STATIC_ASSERT_V(size<1>(tCrA) == size<1>(acc));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<1>(tCrB) == size<2>(acc));                     // MMA_N
    CUTE_STATIC_ASSERT_V(size<2>(tCrA) == size<2>(tCrB));                     // MMA_K
    Tensor tCrA_copy_view = smem_thr_copy_A.retile_D(tCrA);
    CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCrA_copy_view));            // M
    Tensor tCrB_copy_view = smem_thr_copy_B.retile_D(tCrB);
    CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));            // N
    // NOTE: s -> reg
    cute::copy(smem_tiled_copy_A, tCsA(_, _, _0{}), tCrA_copy_view(_, _, _0{}));
    cute::copy(smem_tiled_copy_B, tCsB(_, _, _0{}), tCrB_copy_view(_, _, _0{}));
    #pragma unroll
    for (int i = 0; i < size<2>(tCrA); ++i) {
        if (i < size<2>(tCrA) - 1) {
            cute::copy(smem_tiled_copy_A, tCsA(_, _, i + 1), tCrA_copy_view(_, _, i + 1));
            cute::copy(smem_tiled_copy_B, tCsB(_, _, i + 1), tCrB_copy_view(_, _, i + 1));
        }
        cute::gemm(tiled_mma, tCrA(_, _, i), tCrB(_, _, i), acc);
    }
}

// Blocks until all but N previous cp.async.commit_group operations have committed.
// This differs from cute::cp_async_wait in that when N = 0 we don't call cp.async.wait_all
// (which is equivalent to commit_group then wait_group 0).
// Instead we just call cp.async.wait_group 0, which is slightly faster.
// https://github.com/NVIDIA/cutlass/blob/master/include/cute/arch/copy_sm80.hpp#L113
template <int N>
CUTE_HOST_DEVICE
void cp_async_wait() {
#if defined(CUTE_ARCH_CP_ASYNC_SM80_ENABLED)
    asm volatile("cp.async.wait_group %0;\n" :: "n"(N));
#endif
}

// copy from S to D with tiled_copy
// TODO: 需要支持causal模式的的跳过拷贝
template <typename TiledCopy, typename Engine0, typename Layout0, typename Engine1, typename Layout1>
inline __device__ void copy(TiledCopy tiled_copy, Tensor<Engine0, Layout0> const &S,
                            Tensor<Engine1, Layout1> &D) {
    CUTE_STATIC_ASSERT_V(rank(S) == Int<3>{});
    CUTE_STATIC_ASSERT_V(rank(D) == Int<3>{});
    CUTE_STATIC_ASSERT_V(size<0>(S) == size<0>(D));                     // MMA
    CUTE_STATIC_ASSERT_V(size<1>(S) == size<1>(D));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<2>(S) == size<2>(D));                     // MMA_K

    #pragma unroll
    for (int m = 0; m < size<1>(S); ++m) {
        // TODO: 原版处这里identity_MN是用来跳过大块的block的, predicate用于跳过block内的拷贝
        // TODO: 添加predicate逻辑, 用于跳过无用拷贝
        // if (get<0>(identity_MN(0, m, 0)) < max_MN)
        #pragma unroll
        for (int k = 0; k < size<2>(S); ++k) {
          cute::copy(tiled_copy, S(_, m, k), D(_, m, k));
        }
    }
}


// Convert rowcol_layout from (nrow=(2, MMA_M), ncol=(2, MMA_N)) to ((2, 2, 2), MMA_M, MMA_N / 2)
// if using m16n8k16, or to ((2, 2, 1), MMA_M, MMA_N) if using m16n8k8.
template<typename MMA_traits, typename Layout>
inline __device__ auto convert_layout_rowcol_Aregs(Layout rowcol_layout) {
    using X = Underscore;
    static_assert(decltype(size<0, 0>(rowcol_layout))::value == 2);
    static_assert(decltype(size<1, 0>(rowcol_layout))::value == 2);
    constexpr int mma_shape_K = get<2>(typename MMA_traits::Shape_MNK{});
    static_assert(mma_shape_K == 8 || mma_shape_K == 16);
    constexpr int MMA_N_divisor = mma_shape_K == 8 ? 1 : 2;
    auto l = logical_divide(rowcol_layout, Shape<X, Shape<X, Int<MMA_N_divisor>>>{});  // ((2, MMA_M), (2, (2, MMA_N / 2)))
    // TD [2023-08-13]: Same error as above on Cutlass 3.2
    // return make_layout(make_layout(get<1, 0>(l), get<0, 0>(l), get<1, 1, 0>(l)),
    //                    get<0, 1>(l),
    //                    get<1, 1, 1>(l));
    return make_layout(make_layout(get<0>(get<1>(l)), get<0>(get<0>(l)), get<0>(get<1>(get<1>(l)))),
                       get<1>(get<0>(l)),
                       get<1>(get<1>(get<1>(l))));
};


// TODO: not work
template <typename To_type, typename Engine, typename Layout>
inline __device__ auto convert_type(Tensor<Engine, Layout> const &tensor) {
    using From_type = typename Engine::value_type;
    constexpr int numel = decltype(size(tensor))::value;
    cutlass::NumericArrayConverter<To_type, From_type, numel> convert_op;
    // HACK: this requires tensor to be "contiguous"
    auto frag = convert_op(*reinterpret_cast<const cutlass::Array<From_type, numel> *>(tensor.data()));
    return make_tensor(make_rmem_ptr<To_type>(&frag), tensor.layout());
}

// TODO:
// https://github.com/NVIDIA/cutlass/issues/802
// TODO: convert出来后数据是否在寄存器?
template <typename Fragment>
inline __device__ auto convert_type_f32_to_f16(Fragment const &acc_fp32) {
  Tensor acc_fp16 = make_tensor<cute::half_t>(shape(acc_fp32));
  {
    Tensor acc_fp32x2 = recast< float2>(acc_fp32);
    Tensor acc_fp16x2 = recast<__half2>(acc_fp16);
    for (int i = 0; i < size(acc_fp32x2); ++i) { acc_fp16x2(i) = __float22half2_rn(acc_fp32x2(i)); }
  }
  return acc_fp16;
}

// Apply the exp to all the elements.
template <bool Scale_max=true, typename Engine0, typename Layout0, typename Engine1, typename Layout1>
inline __device__ void scale_apply_exp2(Tensor<Engine0, Layout0> &tensor, Tensor<Engine1, Layout1> const &max, const float scale) {
    static_assert(Layout0::rank == 2, "Only support 2D Tensor");
    static_assert(Layout1::rank == 1, "Only support 1D Tensor");
    CUTE_STATIC_ASSERT_V(size<0>(max) == size<0>(tensor));
    #pragma unroll
    for (int mi = 0; mi < size<0>(tensor); ++mi) {
        // If max is -inf, then all elements must have been -inf (possibly due to masking).
        // We don't want (-inf - (-inf)) since that would give NaN.
        // If we don't have float around M_LOG2E the multiplication is done in fp64.
        const float max_scaled = max(mi) == -INFINITY ? 0.f : max(mi) * (Scale_max ? scale : float(M_LOG2E));
        #pragma unroll
        for (int ni = 0; ni < size<1>(tensor); ++ni)  {
            // Instead of computing exp(x - max), we compute exp2(x * log_2(e) -
            // max * log_2(e)) This allows the compiler to use the ffma
            // instruction instead of fadd and fmul separately.
            tensor(mi, ni) = exp2f(tensor(mi, ni) * scale - max_scaled);
        }
    }
}



// Convert acc_layout from (MMA=4, MMA_M, MMA_N) to (nrow=(2, MMA_M), ncol=(2, MMA_N))
// TODO: 搞清楚经过convert_layout_acc_rowcol后(nrow=(2, MMA_M), ncol=(2, MMA_N))的数学含义
// 形象的解释是把
//    T1.V0
//    T1.V1
//    T1.V0
//    T1.V1
// 变为
//    T1.V0 T1.V1
//    T1.V0 T1.V1
// 这样符合MMA tile的行列直觉
template<typename Layout>
inline __device__ auto convert_layout_acc_rowcol(Layout acc_layout) {
    static_assert(decltype(size<0>(acc_layout))::value == 4);
    static_assert(decltype(rank(acc_layout))::value == 3);
    auto l = logical_divide(acc_layout, Shape<_2>{});  // ((2, 2), MMA_M, MMA_N)
    // TD [2023-08-13]: Idk why but get<0, 1>(l) doesn't work for Cutlass 3.2, I'm getting
    // "int_tuple.hpp(74): error: conversion to inaccessible base class"
    // return make_layout(make_layout(get<0, 1>(l), get<1>(l)), make_layout(get<0, 0>(l), get<2>(l)));
    return make_layout(make_layout(get<1>(get<0>(l)), get<1>(l)), make_layout(get<0>(get<0>(l)), get<2>(l)));
};

// TODO: is first优化初次rescale
template<bool Is_first, typename Tensor0, typename Tensor1, typename Tensor2>
inline __device__ void softmax_rescale_o(Tensor0 &scores, Tensor1 &scores_max, Tensor1 &scores_sum,
                                         Tensor2 &acc_o, float softmax_scale_log2) {
    // NOTE: scores来自acc_s: Q@K.T
    // acc_s用来存储QK和softmax的结果[seqlen, seqlen]
    // acc_o用来存储softmax(QK)结果的分子部分, 用于rescale
    // 流式计算不断用当前分块计算的结果scors来rescale

    if (Is_first) {
        // NOTE: 优化, 第一次softmax不需要rescale, 只需要记录分子, max, sum
        reduce_max</*zero_init=*/true>(scores, scores_max);
        flash::scale_apply_exp2(scores, scores_max, softmax_scale_log2);
        reduce_sum(scores, scores_sum);
    } else {
        // 记录上一次的max
        Tensor scores_max_prev = make_fragment_like(scores_max);
        cute::copy(scores_max, scores_max_prev);
        // TODO: reduce的实现学习一下
        // NOTE: 计算新max到scores_max
        // reduce_max包含步:
        //  1. 求当前thread内max: 遍历
        //  2. reduce thread间的max: 使用shift技巧reduce
        reduce_max</*zero_init=*/false>(scores, scores_max);
        // Reshape acc_o from (MMA=4, MMA_M, MMA_K) to (nrow=(2, MMA_M), ncol=(2, MMA_K))
        // 将acc_o转换成符合2D直觉的(nrow, ncol)的形状
        Tensor acc_o_rowcol = make_tensor(acc_o.data(), flash::convert_layout_acc_rowcol(acc_o.layout()));
        #pragma unroll
        for (int mi = 0; mi < size(scores_max); ++mi) {
            // NOTE: 辅助变量: 当前max
            float scores_max_cur = scores_max(mi);
            // NOTE: 计算旧score的rescale值
            // NOTE: 因为QK(影响max)计算时没有考虑softmax_scale, 所以这里要补上
            float scores_scale = exp2f((scores_max_prev(mi) - scores_max_cur) * softmax_scale_log2);
            // NOTE: rescale旧分母部分
            scores_sum(mi) *= scores_scale;
            // NOTE: 旧分子部分rescale
            // acc_o_rowcol.shape = (nrow, ncol)
            #pragma unroll
            for (int ni = 0; ni < size<1>(acc_o_rowcol); ++ni) { acc_o_rowcol(mi, ni) *= scores_scale; }
        }
        // NOTE: 计算新分子部分: 对所有scores进行rescale
        flash::scale_apply_exp2(scores, scores_max, softmax_scale_log2);

        // NOTE: 累加新分母
        Tensor scores_sum_cur = make_fragment_like(scores_sum);
        // NOTE:利用新分子来累加新分母
        //  1. 线程内累加: 遍历
        //  2. 线程间累加: 使用shift技巧reduce
        reduce_sum(scores, scores_sum_cur);
        // NOTE: 新分母累加到旧分母
        #pragma unroll
        for (int mi = 0; mi < size(scores_sum); ++mi) { scores_sum(mi) += scores_sum_cur(mi); }
    }
};

} // namespace flash

template <typename Kernel_traits, bool Is_causal=false, typename Params>
__global__ void flash_attention_v2_cutlass_kernel(const Params params) {

  using namespace cute;

  // num_m_block: seqlen group
  const int m_block = blockIdx.x;

  // bs * head
  const int base_id = blockIdx.y;
  // The thread index.
  const int tidx = threadIdx.x;
  const int bs_head_offset = base_id * params.head_stride;

  // TODO: 传入泛型
  // NOTE: 小技巧
  using Element = typename Kernel_traits::Element;
  using ElementAccum = typename Kernel_traits::ElementAccum;
  // using TiledMMA = typename Kernel_traits::MMA;
  using TiledMMA = typename Kernel_traits::TiledMma;
  using index_t = typename Kernel_traits::index_t;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutVt = typename Kernel_traits::SmemLayoutVtransposed;
  using SmemLayoutVtNoSwizzle = typename Kernel_traits::SmemLayoutVtransposedNoSwizzle;

  constexpr int kNWarps = Kernel_traits::kNWarps;
  constexpr int kBlockM = Kernel_traits::kBlockM;
  constexpr int kBlockN = Kernel_traits::kBlockN;
  constexpr int kHeadDim = Kernel_traits::kHeadDim;

  // Shared memory.
  extern __shared__ char smem_[];
  using SharedStorage = SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>;
  SharedStorage &shared_storage = *reinterpret_cast<SharedStorage *>(smem_);

  // TODO: base offset for MHA
  // NOTE: convert C pointer to Tensor for convenience
  Tensor Q = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.q_ptr) + bs_head_offset),
      make_shape(params.seqlen, params.dim),
      make_stride(params.dim, Int<1>{}));
  Tensor K = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.k_ptr) + bs_head_offset),
      make_shape(params.seqlen, params.dim),
      make_stride(params.dim, Int<1>{}));
  Tensor V = make_tensor(
      make_gmem_ptr(reinterpret_cast<Element *>(params.v_ptr) + bs_head_offset),
      make_shape(params.seqlen, params.dim),
      make_stride(params.dim, Int<1>{}));

  // 加载Q, K, V分块
  // (kBlockM, kHeadDim, num_tile_n)
  Tensor gQ = local_tile(Q, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // (kBlockN, kHeadDim, num_tile_n)
  // NOTE: loading流水线, 初次加载所需K, V
  Tensor gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(0, _));
  Tensor gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(0, _));

  // 获取MMA抽象
  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(tidx);

  // Construct SMEM tensors.
  Tensor sQ = make_tensor(make_smem_ptr(shared_storage.smem_q.data()), SmemLayoutQ{});
  Tensor sK = make_tensor(make_smem_ptr(shared_storage.smem_k.data()), SmemLayoutK{});
  Tensor sV = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutV{});

  // Tensor for V Transpose; used in GEMM-II.
  Tensor sVt = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVt{});
  Tensor sVtNoSwizzle = make_tensor(make_smem_ptr(shared_storage.smem_v.data()), SmemLayoutVtNoSwizzle{});

  // NOTE: copy抽象
  // NOTE: QKV gmem -> smem拷贝的抽象
  typename Kernel_traits::GmemTiledCopyQKV gmem_tiled_copy_QKV;
  auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_thread_slice(tidx);

  // NOTE: 定义gmem -> smem拷贝的src, dst
  Tensor tQgQ = gmem_thr_copy_QKV.partition_S(gQ(_, _, 0));
  Tensor tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
  Tensor tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
  Tensor tKsK = gmem_thr_copy_QKV.partition_D(sK);
  Tensor tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
  Tensor tVsV = gmem_thr_copy_QKV.partition_D(sV);


  // NOTE: 定义smem -> reg拷贝的dst
  // partition_fragment与partition类似, 只是返回的是寄存器表示
  Tensor tSrQ  = thr_mma.partition_fragment_A(sQ); // (MMA,MMA_M,MMA_K)
  Tensor tSrK  = thr_mma.partition_fragment_B(sK); // (MMA,MMA_N,MMA_K)
  Tensor tOrVt  = thr_mma.partition_fragment_B(sVtNoSwizzle); // (MMA, MMA_K,MMA_N)

  //
  // Copy Atom retiling
  //

  // TODO: 理解这里的atom retiling

  // NOTE: 准备拷贝Q, K, V到smem的copy对象
  auto smem_tiled_copy_Q = make_tiled_copy_A(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_Q = smem_tiled_copy_Q.get_thread_slice(tidx);
  Tensor tSsQ = smem_thr_copy_Q.partition_S(sQ);

  auto smem_tiled_copy_K = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
  auto smem_thr_copy_K = smem_tiled_copy_K.get_thread_slice(tidx);
  Tensor tSsK = smem_thr_copy_K.partition_S(sK);

  // TODO: 拷贝时转置
  // NOTE: smem->reg拷贝Vt
  auto smem_tiled_copy_V = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtomTransposed{}, tiled_mma);
  auto smem_thr_copy_V = smem_tiled_copy_V.get_thread_slice(tidx);
  Tensor tOsVt = smem_thr_copy_V.partition_S(sVt);

  // NOTE: 命名规则, t表示to, s/g表示位置(smem, gmem)
  // 从smem加载时做retiling
  // tKgK表示gmem中的K, 用作gmem->smem的src
  // tKsK表示smem中的K, 用作gmem->smem的dst
  // tSsK表示smem中的K, 用作smem->reg的src

  // 流水线加载初始Q, K
  // 加载Q到smem
  flash::copy(gmem_tiled_copy_QKV, tQgQ, tQsQ);
  // 加载K到smem
  flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
  // 开始执行异步拷贝
  cute::cp_async_fence();

  Tensor rAccOut = partition_fragment_C(tiled_mma, Shape<Int<kBlockM>, Int<kHeadDim>>{});

  // step1: slice-k compute QK block
  // Q[BLOCK_M, BLOCK_N] @ K[BLOCK_M, BLOCK_N].T = O[BLOCK_M, BLOCK_M]
  //
  // step2:
  // advance K, V

  // NOTE: K, V分块的数量: 处理的区间
  const int n_block_min = 0;
  // NOTE: 1. mask between N BLOCKs if is causal mode
  int seqlen_start = m_block * kBlockM;
  int seqlen_end = (m_block + 1) * kBlockM;
  int n_block_max = Is_causal ? cute::ceil_div(seqlen_end, kBlockN) : cute::ceil_div(params.seqlen, kBlockN);

  // NOTE: 需要记录的max
  Tensor scores_max = make_tensor<ElementAccum>(Shape<Int<2 * size<1>(rAccOut)>>{});
  // NOTE: 需要记录的denom
  Tensor scores_sum = make_fragment_like(scores_max);

  clear(rAccOut);

  for (int nbi = n_block_min; nbi < n_block_max; nbi++) {
    auto rAccScore = partition_fragment_C(tiled_mma, make_shape(Int<kBlockM>{}, Int<kBlockN>{}));

    clear(rAccScore);

    // 等待Q, K的gmem -> smem拷贝完成, 即Q, K就绪
    // wait<0>表示等待还剩0个未完成
    flash::cp_async_wait<0>();
    __syncthreads();

    // gemm的同时异步加载V
    gV = local_tile(V, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi, _));
    tVgV = gmem_thr_copy_QKV.partition_S(gV(_, _, 0));
    // 异步加载V到smem
    flash::copy(gmem_tiled_copy_QKV, tVgV, tVsV);
    // 发起异步拷贝
    cute::cp_async_fence();

    // O = Q@K.T
    // NOTE: 加载smem中的数据到reg再做gemm, **加载期间执行retile**
    flash::gemm_smem(rAccScore, tSrQ, tSrK, tSsQ, tSsK, tiled_mma, smem_tiled_copy_Q, smem_tiled_copy_K,
        smem_thr_copy_Q, smem_thr_copy_K
    );

    Tensor scores = make_tensor(rAccScore.data(), flash::convert_layout_acc_rowcol(rAccScore.layout()));

    // NOTE: 2. mask within N BLOCKs
    if (Is_causal ==  true && nbi * kBlockN >= seqlen_start) {
      flash::mask_within_nblock<kBlockM, kBlockN, kNWarps>(scores, m_block, nbi);
    }

    // NOTE: 等待V加载完成, 为下个K加载准备初始状态
    flash::cp_async_wait<0>();
    __syncthreads();

    // advance K
    if (nbi != n_block_max - 1) {
      gK = local_tile(K, make_tile(Int<kBlockN>{}, Int<kHeadDim>{}), make_coord(nbi + 1, _));
      tKgK = gmem_thr_copy_QKV.partition_S(gK(_, _, 0));
      flash::copy(gmem_tiled_copy_QKV, tKgK, tKsK);
      cute::cp_async_fence();
    }

    // 计算softmax
    // NOTE: rAccOut记录softmax后所有的分子
    nbi == 0 ? flash::softmax_rescale_o</*Is_first=*/true>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale) :
      flash::softmax_rescale_o</*Is_first=*/false>(scores, scores_max, scores_sum, rAccOut, params.softmax_scale);

    // 实际执行QK @ V
    // (score AKA rAccScore): QK[M, N] @ V[N, dim]
    // NOTE: DABC: F32F16F16F32, convert D type(F32) to A type(F16)
    // TODO: convert_type目前写死
    Tensor rP = flash::convert_type_f32_to_f16(rAccScore);
    // NOTE: Convert from layout C to layout A
    Tensor tOrP = make_tensor(rP.data(), flash::convert_layout_rowcol_Aregs<TiledMMA>(scores.layout()));

    flash::gemm_A_in_regs(rAccOut, tOrP, tOrVt, tOsVt, tiled_mma, smem_tiled_copy_V, smem_thr_copy_V);
  }

  // NOTE: 最后统一除上分母部分
  // Reshape acc_o from (MMA=4, MMA_M, MMA_K) to (nrow=(2, MMA_M), ncol=(2, MMA_K))
  Tensor acc_o_rowcol = make_tensor(rAccOut.data(), flash::convert_layout_acc_rowcol(rAccOut.layout()));
  // for row
  #pragma unroll
  for (int mi = 0; mi < size<0>(acc_o_rowcol); ++mi) {
    float sum = scores_sum(mi);
    float inv_sum = (sum == 0.f || sum != sum) ? 1.f : 1.f / sum;
    float scale = inv_sum;
    // for col
    #pragma unroll
    for (int ni = 0; ni < size<1>(acc_o_rowcol); ++ni) { 
      acc_o_rowcol(mi, ni) *= scale; 
    }
  }

  // Convert acc_o from fp32 to fp16/bf16
  Tensor rO = flash::convert_type_f32_to_f16(rAccOut);
  // 复用sQ的smem做sO的拷出
  Tensor sO = make_tensor(sQ.data(), typename Kernel_traits::SmemLayoutO{});    // (SMEM_M,SMEM_N)

  // Partition sO to match the accumulator partitioning
  // TODO: review
  auto smem_tiled_copy_O = make_tiled_copy_C(typename Kernel_traits::SmemCopyAtomO{}, tiled_mma);
  auto smem_thr_copy_O = smem_tiled_copy_O.get_thread_slice(tidx);
  Tensor taccOrO = smem_thr_copy_O.retile_S(rO);        // ((Atom,AtomNum), MMA_M, MMA_N)
  Tensor taccOsO = smem_thr_copy_O.partition_D(sO);     // ((Atom,AtomNum),PIPE_M,PIPE_N)

  // NOTE: 先拷贝到smem
  cute::copy(smem_tiled_copy_O, taccOrO, taccOsO);

  Tensor O = make_tensor(
      // Use ElementAccum(f32) to debug
#ifdef DEBUG
      make_gmem_ptr(reinterpret_cast<ElementAccum *>(params.out_ptr) + bs_head_offset),
#else
      make_gmem_ptr(reinterpret_cast<Element *>(params.out_ptr) + bs_head_offset),
#endif
      make_shape(params.seqlen, params.dim),
      make_stride(params.dim, Int<1>{}));
  Tensor gO = local_tile(O, make_tile(Int<kBlockM>{}, Int<kHeadDim>{}), make_coord(m_block, _));

  // 创建到smem -> gmem的拷贝
  typename Kernel_traits::GmemTiledCopyO gmem_tiled_copy_O;
  auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(tidx);
  Tensor tOsO = gmem_thr_copy_O.partition_S(sO);        // ((Atom,AtomNum),ATOM_M,ATOM_N)
  Tensor tOgO = gmem_thr_copy_O.partition_D(gO(_, _, 0));

  __syncthreads();

  // NOTE:: 再拷贝到gmem

  // TODO: review, 这里两个copy的作用
  Tensor tOrO = make_tensor<Element>(shape(tOgO));
  cute::copy(gmem_tiled_copy_O, tOsO, tOrO);

  flash::copy(gmem_tiled_copy_O, tOrO, tOgO);
}

void flash_attention_v2_cuda(FPC *Q, FPC *K, FPC *V, FPC_O *O, int bs, int head, int seqlen, int dim) {
  using Kernel_traits = Test_Traits;
  using Element = typename Kernel_traits::Element;
  using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
  using SmemLayoutK = typename Kernel_traits::SmemLayoutKV;
  using SmemLayoutV = typename Kernel_traits::SmemLayoutKV;

  // Q smem size + KV smem size
  constexpr int kSmemSize = Kernel_traits::kSmemSize;

  int bs_stride = head * seqlen * dim;
  int head_stride = seqlen * dim;
  int seqlen_stride = dim;
  int dim_stride = 1;
  // int smem_size = kSmemSize;
  constexpr size_t smem_size = size_t(sizeof(SharedStorage<Element, SmemLayoutQ, SmemLayoutK, SmemLayoutV>));

  Flash_fwd_params params;
  set_params_fprop(params, bs, head, seqlen, dim, bs_stride, head_stride,
                   seqlen_stride, dim_stride, Q, K, V, O, softmax_scale);

  const int num_m_block =
      (params.seqlen + Kernel_traits::kBlockM - 1) / Kernel_traits::kBlockM;

  auto kernel = &flash_attention_v2_cutlass_kernel<Kernel_traits, IS_CAUSAL, Flash_fwd_params>;
  // NOTE: smem过大时需要设置
  if (smem_size >= 48 * 1024) {
      CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
  }

  dim3 grid(num_m_block, params.bs * params.head, 1);
  dim3 block(size(Kernel_traits::kNThreads));

  kernel<<<grid, block, smem_size>>>(params);
  CUDA_CHECK(hipGetLastError());

  hipDeviceSynchronize();
}

void self_attention_run(FP *Q, FP *K, FP *V, FP *O, int m, int n, bool is_causal, float sm_scale = 1) {
  int mBlock = 2;
  assert(m % mBlock == 0 && "mBlock should align");

  FP *sm_o;
  hipMalloc((void **)&sm_o, sizeof(FP) * m * m);

  dim3 qk_block(m / mBlock, 1, 1);
  naive_nrow_gemm<<<1, qk_block>>>(Q, K, sm_o, sm_scale, 0, m, m, n, mBlock);
  hipDeviceSynchronize();

  // causal mask
  if (is_causal == true) {
    // QK[M, M]
    dim3 grid(m / mBlock, 1, 1);
    dim3 block(mBlock, 1, 1);
    causal_mask_qk<<<grid, block>>>(sm_o, m);
  }

  {
    // TODO: test QK only
    FP *h_sm_o = new FP[m * m];
    hipMemcpy(h_sm_o, sm_o, sizeof(FP) * m * m, hipMemcpyDeviceToHost);
    Tensor Self = make_tensor(h_sm_o, make_shape(m, m), make_stride(m, 1));
    auto tile = make_tile(8, 8);
    print("self QK: \n");
    print("%d,%d:\n", TX, TY);
    print_tensor(local_tile(Self, tile, make_coord(TX, TY)));
    // print("0,1:\n");
    // print_tensor(local_tile(Self, tile, make_coord(0, 1)));
    // print("1,0:\n");
    // print_tensor(local_tile(Self, tile, make_coord(1, 0)));
    // print("1,1:\n");
    // print_tensor(local_tile(Self, tile, make_coord(1, 1)));
    free(h_sm_o);
  }

  // QK[M, M]
  dim3 sm_block(m, 1, 1);
  row_softmax<<<1, sm_block>>>(sm_o, sm_o, m);
  hipDeviceSynchronize();
  {
    // TODO: test QK only
    FP *h_sm_o = new FP[m * m];
    hipMemcpy(h_sm_o, sm_o, sizeof(FP) * m * m, hipMemcpyDeviceToHost);
    Tensor Self = make_tensor(h_sm_o, make_shape(m, m), make_stride(m, 1));
    auto tile = make_tile(8, 8);
    auto coor = make_coord(TX, TY);
    Tensor tSelf = local_tile(Self, tile, coor);
    print("self softmax(QK): \n");
    print_tensor(tSelf);
    free(h_sm_o);
  }

  // QK[M, M] @ V[M, N]
  dim3 qkv_block(m / mBlock, 1, 1);
  naive_pv<<<1, qkv_block>>>(sm_o, V, O, m, n, mBlock);
  hipDeviceSynchronize();

  {
    FP *h_sm_o = new FP[m * n];
    hipMemcpy(h_sm_o, O, sizeof(FP) * m * n, hipMemcpyDeviceToHost);
    Tensor Self = make_tensor(h_sm_o, make_shape(m, n), make_stride(n, 1));
    auto tile = make_tile(8, 8);
    auto coor = make_coord(TX, TY);
    print("self O: \n");
    print_tensor(local_tile(Self, tile, make_coord(TX, TY)));
    print("x,1:\n");
    print_tensor(local_tile(Self, tile, make_coord(TX, TY+1)));
    free(h_sm_o);
  }

  hipFree(sm_o);
}

void self_attention_cuda(FP *Q, FP *K, FP *V, FP *O, int bs, int head, int seqlen, int dim, bool is_causal, float sm_scale = 1) {
  int stride = seqlen * dim;
  int total_size = bs * head * seqlen * dim;
  for (int i = 0; i < bs * head; i++) {
    self_attention_run(Q, K, V, O, seqlen, dim, is_causal, sm_scale);
    Q += stride;
    K += stride;
    V += stride;
    O += stride;
  }
}

// naive gemm implement with slice-k
// perform C = aA@B + bC
// A[M, K] x B[K, N] = C[M, N]
// each thread process mblock rows of A
__global__ void naive_nrow_gemm(FP *A, FP *B, FP *C, FP a, FP b,
                                int M, int N, int K, int mBlock) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // each thread process a range of rows
  idx *= mBlock;

  // A[mBlock, K] x B[N, K].T = C[mBlock, N]
  for (int i = idx; i < idx + mBlock; i++) {
    for (int j = 0; j < N; j++) {
      FP sum = 0.f;
      for (int k = 0; k < K; k++) {
        sum += A[i * K + k] * B[j * K + k];
      }
      // C[M, N]
      // C = aA@B + bC
      C[i * N + j] = a * sum + b * C[i * N + j];
    }
  }
}

__global__ void causal_mask_qk(FP *qk, int m) {
  // each thread process a row
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  int row = idx;
  for (int j = row + 1; j < m; j++) {
    qk[row * m + j] = -INFINITY;
  }
}

// perform QK[M, M] @ V[M, N]
__global__ void naive_pv(FP *P, FP *V, FP *O, int M, int N,
                         int mBlock) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // each thread process a range of rows
  idx *= mBlock;

  int K = M;
  // P[mBlock, M] x V[M, N] = O[mBlock, N]
  for (int i = idx; i < idx + mBlock; i++) {
    for (int j = 0; j < N; j++) {
      FP sum = 0.f;
      for (int k = 0; k < K; k++) {
        sum += P[i * K + k] * V[k * N + j];
      }
      // C[M, N]
      O[i * N + j] = sum;
    }
  }
}

// each thread process one row of softmax
__global__ void row_softmax(FP *input, FP *output, int n) {
  // assume id will not exceed row number of input
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  FP max = -INFINITY;
  FP sum = 0.f;

  // Find max
  for (int i = 0; i < n; i++) {
    if (input[idx * n + i] > max) {
      max = input[idx * n + i];
    }
  }

  // Compute numerator and denominator
  for (int i = 0; i < n; i++) {
    output[idx * n + i] = exp2(input[idx * n + i] - max);
    sum += output[idx * n + i];
  }

  // Compute softmax
  for (int i = 0; i < n; i++) {
    output[idx * n + i] /= sum;
  }
}

void test_attention() {
  int bs = BS;
  int head = HEAD;
  // seqlen
  int m = SEQLEN;
  // dim
  int n = DIM;
  int total_size = bs * head * m * n;

  // Host pointer
  FP *h_K = new FP[total_size];
  FP *h_Q = new FP[total_size];
  FP *h_V = new FP[total_size];
  FP *h_O = new FP[total_size];

  FPC *h_K2 = new FPC[total_size];
  FPC *h_Q2 = new FPC[total_size];
  FPC *h_V2 = new FPC[total_size];
  FPC_O *h_O2 = new FPC_O[total_size];

  // 初始化 K, Q, V
  for (int i = 0; i < total_size; ++i) {
    h_K[i] = static_cast<FP>(rand()) / RAND_MAX;
    h_Q[i] = static_cast<FP>(rand()) / RAND_MAX;
    h_V[i] = static_cast<FP>(rand()) / RAND_MAX;
    // h_K[i] = static_cast<FP>(0.0001f * i);
    // h_Q[i] = static_cast<FP>(0.0001f * i);
    // h_V[i] = static_cast<FP>(0.0001f * i);

    h_Q2[i] = FPC(h_Q[i]);
    h_K2[i] = FPC(h_K[i]);
    h_V2[i] = FPC(h_V[i]);
  }

  FP *d_K, *d_Q, *d_V, *d_O;
  FPC *d_K2, *d_Q2, *d_V2;
  FPC_O *d_O2;
  // Malloc device memory
  hipMalloc((void **)&d_K, sizeof(FP) * total_size);
  hipMalloc((void **)&d_Q, sizeof(FP) * total_size);
  hipMalloc((void **)&d_V, sizeof(FP) * total_size);
  hipMalloc((void **)&d_O, sizeof(FP) * total_size);

  hipMalloc((void **)&d_K2, sizeof(FPC) * total_size);
  hipMalloc((void **)&d_Q2, sizeof(FPC) * total_size);
  hipMalloc((void **)&d_V2, sizeof(FPC) * total_size);
  hipMalloc((void **)&d_O2, sizeof(FPC_O) * total_size);

  // Copy data from host to device
  hipMemcpy(d_K, h_K, sizeof(FP) * total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_Q, h_Q, sizeof(FP) * total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_V, h_V, sizeof(FP) * total_size, hipMemcpyHostToDevice);

  hipMemcpy(d_K2, h_K2, sizeof(FPC) * total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_Q2, h_Q2, sizeof(FPC) * total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_V2, h_V2, sizeof(FPC) * total_size, hipMemcpyHostToDevice);


  // Run test
  for (int i = 0; i < 1; i++) {
    // Launch kernel
    bool is_causal = IS_CAUSAL;
    self_attention_cuda(d_Q, d_K, d_V, d_O, bs, head, m, n, is_causal, softmax_scale);

    CUDA_CHECK(hipGetLastError());
  }
  hipDeviceSynchronize();

  int epoch = 100;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // test flash attention 2
  for (int i = 0; i < epoch; i++) {
    flash_attention_v2_cuda(d_Q2, d_K2, d_V2, d_O2, bs, head, m, n);
    CUDA_CHECK(hipGetLastError());
  }
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for kernel execution: %.3f ms \n", milliseconds / 100);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Result back to host
  hipMemcpy(h_O, d_O, sizeof(FP) * total_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_O2, d_O2, sizeof(FPC_O) * total_size, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  Tensor Cute = make_tensor(h_O2, make_shape(m, n), make_stride(n, 1));
  auto tile = make_tile(8, 8);
  print("cute: \n");
  print_tensor(local_tile(Cute, tile, make_coord(TX, TY)));
  print_tensor(local_tile(Cute, tile, make_coord(TX, TY + 1)));

  assert(all_close(h_O, h_O2, total_size) && "flash attention 1 != flash attention 2");



  hipFree(d_K);
  hipFree(d_Q);
  hipFree(d_V);
  hipFree(d_O);
  hipFree(d_K2);
  hipFree(d_Q2);
  hipFree(d_V2);
  hipFree(d_O2);
  free(h_Q);
  free(h_K);
  free(h_V);
  free(h_O);
  free(h_Q2);
  free(h_K2);
  free(h_V2);
  free(h_O2);
}

template <typename T, typename U>
bool all_close(T *A, U *B, int total_size) {
  for (int i = 0; i < total_size; i++) {
    if (fabs(A[i] - B[i]) > 1e-2) {
      printf("A[%d] = %f, B[%d] = %f\n", i, A[i], i, (float)B[i]);
      return false;
    }
  }
  return true;
}

int main() {
  int epoch = 1;
  for (int i = 0; i < epoch; i++)
    test_attention();

  return 0;
}

